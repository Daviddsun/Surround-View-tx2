#include "hip/hip_runtime.h"
#include "yuv2rgb.cuh"

__device__ inline float clamp(float val, float min, float max)
{
	return (val >= min) ? ((val <= max) ? val : max) : min;
}


/*
__global__ inline void gpuConvertUYVY2RGB_kernel(uchar* src, uchar* dst, uint width, uint height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width)
		return;

	for(int i=0; i<height; ++i){
		int cb = src[i*width*2+idx*4];
		int y0 = src[i*width*2+idx*4+1];
		int cr = src[i*width*2+idx*4+2];
		int y1 = src[i*width*2+idx*4+3];
		
		dst[i*width*3+idx*6]   = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128),                       .0f, 255.f);	
		dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), .0f, 255.f);
		dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), .0f, 255.f);	
					
		dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128),                       .0f, 255.f);
		dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), .0f, 255.f);
		dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), .0f, 255.f);
	}
	
}*/

__global__ inline void gpuConvertUYVY2RGB_kernel(uchar* src, uchar* dst, uint width, uint height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx*2 >= width)
		return;

	for(int y = 0; y < height; ++y){
		int cb = src[y*width*2+idx*4];
		int y0 = src[y*width*2+idx*4+1];
		int cr = src[y*width*2+idx*4+2];
		int y1 = src[y*width*2+idx*4+3];
		
		dst[y*width*3+idx*6]   = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), .0f, 255.f);
		dst[y*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), .0f, 255.f);
		dst[y*width*3+idx*6+2] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128),                       .0f, 255.f);
					
		dst[y*width*3+idx*6+3] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), .0f, 255.f);
		dst[y*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), .0f, 255.f);
		dst[y*width*3+idx*6+5] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128),                       .0f, 255.f);
	}
	
}


void gpuConvertUYVY2RGB(uchar* src, uchar* dst, uint width, uint height)
{

	uchar* d_src = NULL;
	uchar* d_dst = NULL;
	size_t planeSize = width * height * sizeof(uchar);

	uint flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped){
		d_src = src;
		hipStreamAttachMemAsync(NULL, src, 0, hipMemAttachGlobal);
	}
	else{
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped){
		d_dst = dst;
		hipStreamAttachMemAsync(NULL, dst, 0, hipMemAttachGlobal);
	}
	else
		hipMalloc(&d_dst, planeSize * 3);


	uint blockSize = 1024;
	uint numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertUYVY2RGB_kernel <<<numBlocks, blockSize >>>(d_src, d_dst, width, height);
	hipStreamAttachMemAsync(NULL, dst, 0 , hipMemAttachHost); // Host?
	hipStreamSynchronize(NULL);
	if (!srcIsMapped){
		hipMemcpy(dst, d_dst, planeSize*3, hipMemcpyDeviceToHost);
		hipFree(d_src);
	}
	if (!dstIsMapped)
		hipFree(d_dst);

}



void gpuConvertUYVY2RGB_async(uchar* src, uchar* dst, uint width, uint height, hipStream_t stream)
{

	uchar* d_src = NULL;
	uchar* d_dst = NULL;
	size_t planeSize = width * height * sizeof(uchar);

	uint flags;
	bool srcIsMapped = (hipHostGetFlags(&flags, src) == hipSuccess) && (flags & hipHostMallocMapped);
	bool dstIsMapped = (hipHostGetFlags(&flags, dst) == hipSuccess) && (flags & hipHostMallocMapped);

	if (srcIsMapped){
		d_src = src;
		hipStreamAttachMemAsync(stream, src, 0, hipMemAttachGlobal);
	}
	else{
		hipMalloc(&d_src, planeSize * 2);
		hipMemcpy(d_src, src, planeSize * 2, hipMemcpyHostToDevice);
	}
	if (dstIsMapped){
		d_dst = dst;
		hipStreamAttachMemAsync(stream, dst, 0, hipMemAttachGlobal);
	}
	else
		hipMalloc(&d_dst, planeSize * 3);

	uint blockSize = 1024;
	uint numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertUYVY2RGB_kernel <<<numBlocks, blockSize >>>(d_src, d_dst, width, height);

	hipStreamAttachMemAsync(stream, dst, 0 , hipMemAttachGlobal);
	//hipStreamSynchronize(NULL);

	if (!srcIsMapped){
		hipMemcpy(dst, d_dst, planeSize*3, hipMemcpyDeviceToHost);
		hipFree(d_src);
	}
	if (!dstIsMapped)
		hipFree(d_dst);

}



